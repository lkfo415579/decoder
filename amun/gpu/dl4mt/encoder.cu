#include "hip/hip_runtime.h"
#include "encoder.h"
#include "common/sentences.h"

using namespace std;

namespace amunmt {
namespace GPU {

Encoder::Encoder(const Weights& model, const YAML::Node& config)
  : embeddings_(model.encEmbeddings_),
    forwardRnn_(InitForwardCell(model, config)),
    backwardRnn_(InitBackwardCell(model, config))
{}

std::unique_ptr<Cell> Encoder::InitForwardCell(const Weights& model, const YAML::Node& config){
  std::string celltype = config["enc-cell"] ? config["enc-cell"].as<std::string>() : "gru";
  if (celltype == "lstm") {
    return unique_ptr<Cell>(new LSTM<Weights::EncForwardLSTM>(*(model.encForwardLSTM_)));
  } else if (celltype == "mlstm") {
    return unique_ptr<Cell>(new Multiplicative<LSTM, Weights::EncForwardLSTM>(*model.encForwardMLSTM_));
  } else if (celltype == "gru") {
    return unique_ptr<Cell>(new GRU<Weights::EncForwardGRU>(*(model.encForwardGRU_)));
  }

  assert(false);
  return unique_ptr<Cell>(nullptr);
}

std::unique_ptr<Cell> Encoder::InitBackwardCell(const Weights& model, const YAML::Node& config){
  std::string enccell = config["enc-cell"] ? config["enc-cell"].as<std::string>() : "gru";
  std::string celltype = config["enc-cell-r"] ? config["enc-cell-r"].as<std::string>() : enccell;
  if (celltype == "lstm") {
    return unique_ptr<Cell>(new LSTM<Weights::EncBackwardLSTM>(*(model.encBackwardLSTM_)));
  } else if (celltype == "mlstm") {
    return unique_ptr<Cell>(new Multiplicative<LSTM, Weights::EncBackwardLSTM>(*model.encBackwardMLSTM_));
  } else if (celltype == "gru") {
    return unique_ptr<Cell>(new GRU<Weights::EncBackwardGRU>(*(model.encBackwardGRU_)));
  }

  assert(false);
  return unique_ptr<Cell>(nullptr);
}

size_t GetMaxLength(const Sentences& source, size_t tab) {
  size_t maxLength = source.at(0)->GetWords(tab).size();
  for (size_t i = 0; i < source.size(); ++i) {
    const Sentence &sentence = *source.at(i);
    maxLength = std::max(maxLength, sentence.GetWords(tab).size());
  }
  return maxLength;
}

std::vector<std::vector<uint>> GetBatchInput(const Sentences& source, size_t tab, size_t maxLen) {
  std::vector<std::vector<uint>> matrix(maxLen, std::vector<uint>(source.size(), 0));

  for (size_t j = 0; j < source.size(); ++j) {
    for (size_t i = 0; i < source.at(j)->GetWords(tab).size(); ++i) {
        matrix[i][j] = source.at(j)->GetWords(tab)[i];
    }
  }

  return matrix;
}

void Encoder::Encode(const Sentences& source, size_t tab, mblas::Matrix& context,
                         mblas::Vector<uint> &sentenceLengths)
{
  size_t maxSentenceLength = GetMaxLength(source, tab);

  std::vector<uint> hSentenceLengths(source.size());
  for (size_t i = 0; i < source.size(); ++i) {
    hSentenceLengths[i] = source.at(i)->GetWords(tab).size();
  }

  sentenceLengths.newSize(source.size());
  mblas::copy(hSentenceLengths.data(),
              hSentenceLengths.size(),
              sentenceLengths.data(),
              hipMemcpyHostToDevice);

  //cerr << "GetContext1=" << context.Debug(1) << endl;
  context.NewSize(maxSentenceLength,
                 forwardRnn_.GetStateLength().output + backwardRnn_.GetStateLength().output,
                 1,
                 source.size());
  //cerr << "GetContext2=" << context.Debug(1) << endl;

  auto input = GetBatchInput(source, tab, maxSentenceLength);

  for (size_t i = 0; i < input.size(); ++i) {
    if (i >= embeddedWords_.size()) {
      embeddedWords_.emplace_back();
    }
    embeddings_.Lookup(embeddedWords_[i], input[i]);
    //cerr << "embeddedWords_=" << embeddedWords_.back().Debug(true) << endl;
  }

  //cerr << "GetContext3=" << context.Debug(1) << endl;
  forwardRnn_.Encode(embeddedWords_.cbegin(),
                         embeddedWords_.cbegin() + maxSentenceLength,
                         context, source.size(), false);
  //cerr << "GetContext4=" << context.Debug(1) << endl;

  backwardRnn_.Encode(embeddedWords_.crend() - maxSentenceLength,
                          embeddedWords_.crend() ,
                          context, source.size(), true, &sentenceLengths);
  //cerr << "GetContext5=" << context.Debug(1) << endl;
}

}
}

