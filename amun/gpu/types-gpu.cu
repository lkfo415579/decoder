#include <iostream>
#include "types-gpu.h"
#include "mblas/handles.h"

namespace amunmt {
namespace GPU {

void HandleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    std::cerr << "ERROR: " << hipGetErrorString(err) << " in " << file << " at line " << line << std::endl;
    exit( EXIT_FAILURE );
  }
}


}
}
