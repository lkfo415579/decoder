#include <iostream>
#include "common/utils.h"
#include "matrix_wrapper.h"
#include "vector_wrapper.h"
#include "nth_element.h"
#include "matrix_functions.h"

using namespace std;

namespace amunmt {
namespace GPU {

NthElement::NthElement(uint maxBeamSize, uint maxBatchSize)
: d_breakdown(maxBeamSize)
, maxBeamSize_(maxBeamSize)
, maxBatchSize_(maxBatchSize)
{
  //cerr << "maxBatchSize=" << maxBatchSize << " maxBeamSize=" << maxBeamSize << endl;

  d_batchPosition.reserve(maxBatchSize + 1);
  d_cumBeamSizes.reserve(maxBatchSize + 1);

  d_res.reserve(maxBatchSize * maxBeamSize);
  h_res.reserve(maxBatchSize * maxBeamSize);
}

NthElement::~NthElement()
{
  //cerr << "FOO2" << endl;
}

void NthElement::getNBestList(const std::vector<uint>& beamSizes, mblas::Matrix& Probs,
                  std::vector<float>& outCosts, std::vector<uint>& outKeys,
                  const bool isFirst) {
  /*
  cerr << "beamSizes=" << beamSizes.size() << endl;
  cerr << Debug(beamSizes, 2) << endl;
  cerr << "Probs=" << Probs.Debug(0) << endl;
  cerr << "outCosts=" << outCosts.size() << endl;
  cerr << "outKeys=" << outKeys.size() << endl;
  cerr << "isFirst=" << isFirst << endl;
  cerr << endl;
  */
  std::vector<uint> cummulatedBeamSizes(beamSizes.size() + 1);
  std::vector<uint> batchFirstElementIdxs(beamSizes.size() + 1);
  cummulatedBeamSizes[0] = 0;
  batchFirstElementIdxs[0] = 0;

  const uint vocabSize = Probs.dim(1);
  for (uint i = 0; i < beamSizes.size(); ++i) {

    cummulatedBeamSizes[i + 1] = cummulatedBeamSizes[i] + beamSizes[i];
    batchFirstElementIdxs[i + 1] = ((isFirst) ? (i + 1) : cummulatedBeamSizes[i + 1]) * vocabSize;
  }

  uint numHypos = cummulatedBeamSizes.back();
  d_res.newSize(numHypos);
  h_res.resize(numHypos);

  //cerr << endl;
  //cerr << "numHypos=" << numHypos << endl;
  //cerr << "beamSizes=" << Debug(beamSizes, 2) << endl;
  //cerr << "cummulatedBeamSizes=" << Debug(cummulatedBeamSizes, 2) << endl;
  //cerr << "batchFirstElementIdxs=" << Debug(batchFirstElementIdxs, 2) << endl;
  //cerr << "1Probs=" << Probs.Debug() << endl;

  getNBestList(Probs, batchFirstElementIdxs, cummulatedBeamSizes);

  //cerr << "2Probs=" << Probs.Debug() << endl;
  //cerr << "cummulatedBeamSizes.back()=" << cummulatedBeamSizes.back() << endl;
  //cerr << "cummulatedBeamSizes=" << Debug(cummulatedBeamSizes, 2) << endl;
  GetPairs(numHypos, outKeys, outCosts);

  //cerr << "outCosts=" << Debug(outCosts, 2) << endl;
  //cerr << "outKeys=" << Debug(outKeys, 2) << endl;
}

void NthElement::getNBestList(mblas::Matrix &probs,
                              const std::vector<uint>& batchFirstElementIdxs,
                              const std::vector<uint>& cummulatedBeamSizes)
{
  const uint vocabSize = probs.dim(1);
  const uint numBlocks = uint(maxBeamSize_ * vocabSize / (2 * BLOCK_SIZE)) + uint(maxBeamSize_ * vocabSize % (2 * BLOCK_SIZE) != 0);
  const uint numBatches = batchFirstElementIdxs.size() - 1;

  d_out.newSize(maxBatchSize_ * numBlocks);

  //cerr << "cummulatedBeamSizes=" << cummulatedBeamSizes.size() << endl;
  d_batchPosition.newSize(batchFirstElementIdxs.size());
  d_cumBeamSizes.newSize(cummulatedBeamSizes.size());
  assert(d_batchPosition.size() == d_cumBeamSizes.size());

  mblas::copy(batchFirstElementIdxs.data(),
              batchFirstElementIdxs.size(),
              d_batchPosition.data(),
              hipMemcpyHostToDevice);
  mblas::copy(cummulatedBeamSizes.data(),
              cummulatedBeamSizes.size(),
              d_cumBeamSizes.data(),
              hipMemcpyHostToDevice);

  mblas::VectorWrapper<NthOut> outWrap(d_out);
  mblas::MatrixWrapper<float> probsWrap(probs);
  mblas::VectorWrapper<uint> batchPositionWrap(d_batchPosition);
  mblas::VectorWrapper<NthOut> resWrap(d_res);
  mblas::VectorWrapper<uint> cumBeamSizesWrap(d_cumBeamSizes);

  gMaxElement<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(float), mblas::CudaStreamHandler::GetStream()>>>
    (outWrap, probsWrap, batchPositionWrap, numBatches);

  gMaxElementUpdate<<<numBatches, BLOCK_SIZE, BLOCK_SIZE * sizeof(float), mblas::CudaStreamHandler::GetStream()>>>
    (outWrap,
     probsWrap,
     resWrap,
     batchPositionWrap,
     cumBeamSizesWrap,
     numBlocks);

  /*
  cerr << "numBlocks=" << numBlocks << endl;
  cerr << "numBatches=" << numBatches << endl;
  cerr << "threads=" << BLOCK_SIZE << endl;

  cerr << "outWrap=" << outWrap.Debug() << endl;

  cerr << "probsWrap=" << probsWrap.Debug() << endl;

  cerr << "batchPositionWrap=" << batchPositionWrap.Debug() << endl;
  cerr << mblas::Debug(d_batchPosition, 2) << endl;

  cerr << "resWrap=" << resWrap.Debug() << endl;
  cerr << mblas::Debug(d_res, 2) << endl;

  cerr << "cumBeamSizesWrap=" << cumBeamSizesWrap.Debug() << endl;
  //cerr << mblas::Debug(d_cumBeamSizes, 2) << endl;

  cerr << endl;
  */
}

void NthElement::GetPairs(uint number,
                    std::vector<uint>& outKeys,
                    std::vector<float>& outValues)
{
  mblas::copy(d_res.data(), d_res.size(), h_res.data(), hipMemcpyDeviceToHost);
  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()) );

  for (uint i = 0; i < number; ++i) {
    outKeys.push_back(h_res[i].ind);
    outValues.push_back(h_res[i].score);
  }
}

void NthElement::getValueByKey(std::vector<float>& out, const mblas::Matrix &d_in) const
{
  // need a model with multiple scorers to test this method
  assert(false);

  mblas::VectorWrapper<float> breakdownWrap(d_breakdown);
  const mblas::MatrixWrapper<float> inWrap(d_in);

  //gGetValueByKey<<<1, lastN_, 0, stream_>>>
  //  (breakdownWrap, inWrap, h_res_idx, lastN_);

  HANDLE_ERROR( hipMemcpyAsync(out.data(), d_breakdown.data(), h_res.size() * sizeof(float),
                                hipMemcpyDeviceToHost, mblas::CudaStreamHandler::GetStream()) );
  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
}

//////////////////////////////////////////////////////////////////////////

}  // namespace GPU
} // namespace amunmt
