#include "best_hyps.h"

namespace amunmt {
namespace GPU {

BestHyps::BestHyps(const God &god)
      : BestHypsBase(god),
        keys_(god.Get<size_t>("beam-size") * god.Get<size_t>("mini-batch")),
        costs_(god.Get<size_t>("beam-size") * god.Get<size_t>("mini-batch")),
        maxBeamSize_(god.Get<uint>("beam-size"))
{
  if (!god_.UseFusedSoftmax()) {
    NthElement *obj = new NthElement(god.Get<size_t>("beam-size"), god.Get<size_t>("mini-batch"));
    nthElement_.reset(obj);
  }
}

void BestHyps::DisAllowUNK(mblas::Matrix& Prob) {
  SetColumn(Prob, UNK_ID, std::numeric_limits<float>::lowest());
}

void BestHyps::FindBests(const std::vector<uint>& beamSizes, mblas::Matrix& Probs,
               std::vector<float>& outCosts,
               std::vector<unsigned>& outKeys,
               const bool isFirst)
{
  nthElement_->getNBestList(beamSizes, Probs, outCosts, outKeys, isFirst);
}

// fast fused softmax and nth_element
void BestHyps::FindBests(const std::vector<uint>& beamSizes, mblas::Matrix& Probs,
               mblas::Vector<NthOutBatch> &nBest,
               std::vector<float>& outCosts,
               std::vector<unsigned>& outKeys,
               const bool isFirst)
{
  getNBestList(beamSizes, Probs, nBest, outCosts, outKeys, isFirst);
}

std::vector<SoftAlignmentPtr> BestHyps::GetAlignments(const std::vector<ScorerPtr>& scorers,
                                            size_t hypIndex)
{
  std::vector<SoftAlignmentPtr> alignments;
  for (auto& scorer : scorers) {
    if (GPU::EncoderDecoder* encdec = dynamic_cast<GPU::EncoderDecoder*>(scorer.get())) {
      const mblas::Matrix &attention = encdec->GetAttention();
      size_t attLength = attention.dim(1);

      SoftAlignment *softAlignment = new SoftAlignment(attLength);
      mblas::copy(
          attention.data() + hypIndex * attLength,
          attLength,
          softAlignment->data(),
          hipMemcpyDeviceToHost
      );

      alignments.emplace_back(softAlignment);
    } else {
      amunmt_UTIL_THROW2("Return Alignment is allowed only with Nematus scorer.");
    }
  }
  return alignments;
}

// standard nth_element
void  BestHyps::CalcBeam(
    const Beam& prevHyps,
    const std::vector<ScorerPtr>& scorers,
    const Words& filterIndices,
    std::vector<Beam>& beams,
    std::vector<uint>& beamSizes)
{
  BEGIN_TIMER("CalcBeam");

  using namespace mblas;

  mblas::Matrix& Probs = static_cast<mblas::Matrix&>(scorers[0]->GetProbs());

  std::vector<float> vCosts;
  for (auto& h : prevHyps) {
    vCosts.push_back(h->GetCost());
  }

  mblas::copy(vCosts.data(),
              vCosts.size(),
              costs_.data(),
              hipMemcpyHostToDevice);
  //mblas::copy(vCosts.begin(), vCosts.end(), costs_.begin());

  size_t beamSizeSum = std::accumulate(beamSizes.begin(), beamSizes.end(), 0);

  std::vector<float> bestCosts;
  std::vector<unsigned> bestKeys;

  const bool isFirst = (vCosts[0] == 0.0f) ? true : false;

  if (god_.UseFusedSoftmax()) {
    const mblas::Matrix& b4 = *static_cast<const mblas::Matrix*>(scorers[0]->GetBias());
    mblas::Vector<NthOutBatch> &nBest = *static_cast<mblas::Vector<NthOutBatch>*>(scorers[0]->GetNBest());
    nBest.newSize(beamSizeSum);

    BEGIN_TIMER("GetProbs.LogSoftmaxAndNBest");
    mblas::LogSoftmaxAndNBest(nBest, Probs, b4, costs_, forbidUNK_, maxBeamSize_, beamSizes, beamSizeSum, isFirst);
    PAUSE_TIMER("GetProbs.LogSoftmaxAndNBest");
    //std::cerr << "2Probs=" << Probs.Debug(1) << std::endl;

    FindBests(beamSizes, Probs, nBest, bestCosts, bestKeys, isFirst);
  }
  else {
    BroadcastVecColumn(weights_.at(scorers[0]->GetName()) * _1 + _2, Probs, costs_);

    for (size_t i = 1; i < scorers.size(); ++i) {
      mblas::Matrix &currProbs = static_cast<mblas::Matrix&>(scorers[i]->GetProbs());

      Element(_1 + weights_.at(scorers[i]->GetName()) * _2, Probs, currProbs);
    }

    if (forbidUNK_) {
      DisAllowUNK(Probs);
    }

    FindBests(beamSizes, Probs, bestCosts, bestKeys, isFirst);
  }

  std::vector<std::vector<float>> breakDowns;
  if (god_.ReturnNBestList()) {
      breakDowns.push_back(bestCosts);
      for (size_t i = 1; i < scorers.size(); ++i) {
        std::vector<float> modelCosts(beamSizeSum);
        mblas::Matrix &currProbs = static_cast<mblas::Matrix&>(scorers[i]->GetProbs());

        nthElement_->getValueByKey(modelCosts, currProbs);
        breakDowns.push_back(modelCosts);
      }
  }

  std::map<size_t, size_t> batchMap;
  size_t tmp = 0;
  for (size_t batchID = 0; batchID < beamSizes.size(); ++batchID) {
    for (size_t t = 0; t < beamSizes[batchID]; ++t) {
      batchMap[tmp++] = batchID;
    }
  }

  for (size_t i = 0; i < beamSizeSum; i++) {
    size_t wordIndex = bestKeys[i] % Probs.dim(1);
    if (isInputFiltered_) {
      wordIndex = filterIndices[wordIndex];
    }

    size_t hypIndex  = bestKeys[i] / Probs.dim(1);
    float cost = bestCosts[i];

    HypothesisPtr hyp;
    if (returnAttentionWeights_) {
      hyp.reset(new Hypothesis(prevHyps[hypIndex], wordIndex, hypIndex, cost,
                               GetAlignments(scorers, hypIndex)));
    } else {
      hyp.reset(new Hypothesis(prevHyps[hypIndex], wordIndex, hypIndex, cost));
    }

    if(god_.ReturnNBestList()) {
      hyp->GetCostBreakdown().resize(scorers.size());
      float sum = 0;
      for (size_t j = 0; j < scorers.size(); ++j) {
        if (j == 0)
          hyp->GetCostBreakdown()[0] = breakDowns[0][i];
        else {
          float cost = 0;
          if (j < scorers.size()) {
              if (prevHyps[hypIndex]->GetCostBreakdown().size() < scorers.size())
                const_cast<HypothesisPtr&>(prevHyps[hypIndex])->GetCostBreakdown().resize(scorers.size(), 0.0f);
              cost = breakDowns[j][i] + const_cast<HypothesisPtr&>(prevHyps[hypIndex])->GetCostBreakdown()[j];
          }
          sum += weights_.at(scorers[j]->GetName()) * cost;
          hyp->GetCostBreakdown()[j] = cost;
        }
      }
      hyp->GetCostBreakdown()[0] -= sum;
      hyp->GetCostBreakdown()[0] /= weights_.at(scorers[0]->GetName());
    }

    beams[batchMap[i]].push_back(hyp);
  }

  PAUSE_TIMER("CalcBeam");
}

//////////////////////////////////////////////////////////////////////////
void BestHyps::getNBestList(const std::vector<uint>& beamSizes,
                  mblas::Matrix& Probs,
                  mblas::Vector<NthOutBatch> &nBest,
                  std::vector<float>& outCosts,
                  std::vector<uint>& outKeys,
                  const bool isFirst) const
{
  GetPairs(nBest, outKeys, outCosts);
  assert(outCosts.size() == outKeys.size());

  /*
  cerr << "outCosts/outKeys=";
  for (size_t i = 0; i < outKeys.size(); ++i) {
    cerr << "(" << outCosts[i] << "," << outKeys[i] << ") ";
  }
  cerr << endl;
  */
  //cerr << endl;
}

void BestHyps::GetPairs(mblas::Vector<NthOutBatch> &nBest,
              std::vector<uint>& outKeys,
              std::vector<float>& outValues) const
{
  //cerr << "top=" << top2.size() << " nBest=" << nBest.size() << endl;
  outKeys.resize(nBest.size());
  outValues.resize(nBest.size());

  std::vector<NthOutBatch> hostVec(nBest.size());
  mblas::copy(nBest.data(), nBest.size(), hostVec.data(), hipMemcpyDeviceToHost);

  for (size_t i = 0; i < nBest.size(); ++i) {
    outKeys[i] = hostVec[i].ind;
    outValues[i] = hostVec[i].score;
  }
}

} // namespace
}
