#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include "kernels/cuda_helpers.h"
#include "kernels/tensor_operators.h"

#include "3rd_party/reduce_all.h"

namespace marian {

#define CUDA_FLT_MAX 1.70141e+38

struct isnan_test {
  __host__ __device__ bool operator()(const float a) const { return isnan(a); }
};

__device__ inline float stableLogit(float x) {
  if(x >= 0) {
    float z = expf(-x);
    return 1.0 / (1.0 + z);
  } else {
    float z = expf(x);
    return z / (1.0 + z);
  }
}

bool IsNan(Tensor in) {
  //hipSetDevice(in->getDevice());
  //thrust::device_ptr<float> begin = thrust::device_pointer_cast(in->data());
  //thrust::device_ptr<float> end
  //    = thrust::device_pointer_cast(in->data() + in->size());
  //return thrust::transform_reduce(
  //    begin, end, isnan_test(), 0, thrust::plus<bool>());
  return false;
}

void ConcatCont(Tensor out, const std::vector<Tensor>& inputs, int axis) {
  hipSetDevice(out->getDevice());
  int step = 1;
  for(int i = 0; i < axis; ++i)
    step *= out->shape()[i];

  size_t offset1 = 0;
  for(int i = 0; i < step; ++i) {
    for(auto in : inputs) {
      size_t size = in->shape().elements() / step;
      size_t offset2 = i * size;

      hipMemcpyAsync(out->data() + offset1,
                      in->data() + offset2,
                      size * sizeof(float),
                      hipMemcpyDeviceToDevice);

      offset1 += size;
    }
  }
  hipStreamSynchronize(0);
}

__global__ void gInsertCols(float* out,
                            const float* in,
                            size_t rows,
                            size_t cols,
                            size_t cols_out,
                            size_t cols_in,
                            size_t offset_out,
                            size_t offset_in) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols_out + offset_out;
      const float* rowIn = in + j * cols_in + offset_in;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

void Concatenate1(Tensor out, const std::vector<Tensor>& inputs) {
  hipSetDevice(out->getDevice());

  int rows = out->shape().elements() / out->shape().back();

  size_t offset = 0;
  int cols_out = out->shape().back();

  for(auto in : inputs) {
    ABORT_IF(rows != in->shape().elements() / in->shape().back(),
                   "First dimension must be equal");
    int cols_in = in->shape().back();


    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols_in);

    gInsertCols<<<blocks, threads>>>(
        out->data(), in->data(), rows, cols_in, cols_out, cols_in, offset, 0);
    offset += cols_in;
  }
  hipStreamSynchronize(0);
}

void Concatenate(Tensor out, const std::vector<Tensor>& inputs, int ax) {
  if(ax == out->shape().size() - 1)
    Concatenate1(out, inputs);
  else
    ConcatCont(out, inputs, ax);
}

void Split1(std::vector<Tensor>& outputs, const Tensor in) {
  hipSetDevice(in->getDevice());

  size_t offset = 0;
  int rows = in->shape().elements() / in->shape().back();
  int cols_in = in->shape().back();
  for(auto out : outputs) {
    ABORT_IF(rows != out->shape().elements() / out->shape().back(),
            "First dimension must be equal");
    int cols_out = out->shape().back();

    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols_out);

    gInsertCols<<<blocks, threads>>>(
        out->data(), in->data(), rows, cols_out, cols_out, cols_in, 0, offset);
    offset += cols_out;
  }
  hipStreamSynchronize(0);
}

void SplitCont(std::vector<Tensor>& outputs, const Tensor in, int axis) {
  hipSetDevice(in->getDevice());

  int step = 1;
  for(int i = 0; i < axis; ++i)
    step *= in->shape()[i];

  size_t offset1 = 0;
  for(int i = 0; i < step; ++i) {
    for(auto out : outputs) {
      size_t size = out->shape().elements() / step;
      size_t offset2 = i * size;

      hipMemcpyAsync(out->data() + offset2,
                      in->data() + offset1,
                      size * sizeof(float),
                      hipMemcpyDeviceToDevice);

      offset1 += size;
    }
  }
  hipStreamSynchronize(0);
}

void Deconcatenate(std::vector<Tensor>& outputs, const Tensor in, int ax) {
  if(ax == in->shape().size() - 1)
    Split1(outputs, in);
  else
    SplitCont(outputs, in, ax);
}

__global__ void gTransposeND(gpu::Tensor<float> out,
                             const gpu::Tensor<float> in,
                             const gpu::Array<int, gpu::Shape::size()> permute) {

  constexpr size_t N = gpu::Shape::size();
  gpu::Array<int, N> oDims;
  gpu::Array<int, N> pDims;

  int length = out.shape().elements();
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      out.shape().dims(index, oDims);
      for(int i = 0; i < N; ++i)
        pDims[permute[i]] = oDims[i];
      out[index] = in[pDims];
    }
  }
}

void TransposeND(Tensor out, Tensor in, const std::vector<int>& vAxis) {
  hipSetDevice(out->getDevice());

  gpu::Array<int, gpu::Shape::size()> axes;
  int diff = gpu::Shape::size() - vAxis.size();
  for(int i = 0; i < axes.size(); ++i)
    if(i < diff)
      axes[i] = i;
    else
      axes[i] = vAxis[i - diff] + diff;

  int length = out->shape().elements();
  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gTransposeND<<<blocks, threads>>>(out, in, axes);
}

__global__ void gSoftmax(float* out,
                         gpu::Shape outShape,
                         const float* in,
                         const float* mask,
                         const gpu::Shape maskShape) {
  int rows = outShape.elements() / outShape.back();
  int cols = outShape.back();

  bool broadcast = outShape != maskShape;
  gpu::Array<int, gpu::Shape::size()> dims;

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      extern __shared__ float _share[];

      float* _max = _share + blockDim.x;
      _max[threadIdx.x] = -CUDA_FLT_MAX;  // mask
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float mVal = 1.f;
          if(mask) {
            int mIndex = id + j * cols;
            if(broadcast) {
              outShape.dims(mIndex, dims);
              mIndex = maskShape.bindex(dims);
            }
            mVal = mask[mIndex];
          }

          if(mVal && sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float mVal = 1.f;
          if(mask) {
            int mIndex = id + j * cols;
            if(broadcast) {
              outShape.dims(mIndex, dims);
              mIndex = maskShape.bindex(dims);
            }
            mVal = mask[mIndex];
          }

          float ex = 0;
          if(mVal)
            ex = __expf(sp[id] - max);
          so[id] = ex;

          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          so[id] = so[id] / _sum[0];
        }
      }
    }
  }
}

void Softmax(Tensor out, Tensor in, Tensor mask) {
  hipSetDevice(out->getDevice());

  size_t m = out->shape().elements() / out->shape().back();
  size_t k = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads * 2;

  if(mask)
    gSoftmax<<<blocks, threads, shared>>>(
        out->data(), out->shape(), in->data(), mask->data(), mask->shape());
  else
    gSoftmax<<<blocks, threads, shared>>>(
        out->data(), out->shape(), in->data(), 0, out->shape());
}

__global__ void gLogSoftmax(float* out,
                            const gpu::Shape outShape,
                            const float* in) {
  int rows = outShape.elements() / outShape.back();
  int cols = outShape.back();

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      extern __shared__ float _share[];

      float* _max = _share + blockDim.x;
      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if(sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float sm = sp[id] - max;
          float ex = __expf(sm);
          so[id] = sm;
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols)
          so[id] -= __logf(_sum[0]);
      }
    }
  }
}

void LogSoftmax(Tensor out, Tensor in) {
  hipSetDevice(out->getDevice());

  size_t m = out->shape().elements() / out->shape().back();
  size_t k = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads * 2;

  gLogSoftmax<<<blocks, threads, shared>>>(
      out->data(), out->shape(), in->data());
}

///////////////////////////////////////////////////////

__global__ void gSoftmaxGrad(float* grad,
                             const float* adj,
                             const float* val,
                             const int rows,
                             const int cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;

      float* gradRow = grad + j * cols;
      const float* adjRow = adj + j * cols;
      const float* valRow = val + j * cols;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += valRow[id] * adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float val = valRow[id] * (adjRow[id] - _sum[0]);
          if(val)
            gradRow[id] += val;
        }
      }
    }
  }
}

void SoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
  hipSetDevice(adj->getDevice());
  // grad and val are both m-by-k matrices, passed as input.
  // A weighted average of each row of grad (according to the weights
  // specified in val) is computed and subtracted from Out.
  // adj is multiplied for each element to get backward step in autodiff
  int m = grad->shape().elements() / grad->shape().back();
  int k = grad->shape().back();


  int blocks = std::min(MAX_BLOCKS, m);
  int threads = std::min(MAX_THREADS, k);
  int shared = sizeof(float) * threads * 2;
  gSoftmaxGrad<<<blocks, threads, shared>>>(
      grad->data(), adj->data(), val->data(), m, k);
}

__global__ void gLogSoftmaxGrad(float* grad,
                                const float* adj,
                                const float* val,
                                const int rows,
                                const int cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;

      float* gradRow = grad + j * cols;
      const float* adjRow = adj + j * cols;
      const float* valRow = val + j * cols;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols)
          gradRow[id] += adjRow[id] - (expf(valRow[id]) * _sum[0]);
      }
    }
  }
}

void LogSoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
  hipSetDevice(adj->getDevice());

  // grad and val are both m-by-k matrices, passed as input.
  // A weighted average of each row of grad (according to the weights
  // specified in val) is computed and subtracted from Out.
  // adj is multiplied for each element to get backward step in autodiff
  int m = grad->shape().elements() / grad->shape().back();
  int k = grad->shape().back();

  int blocks = std::min(MAX_BLOCKS, m);
  int threads = std::min(MAX_THREADS, k);
  int shared = sizeof(float) * threads * 2;
  gLogSoftmaxGrad<<<blocks, threads, shared>>>(
      grad->data(), adj->data(), val->data(), m, k);
}

///////////////////////////////////////////////////////
__global__ void gArgmax(float* out,
                        const float* data,
                        size_t rows,
                        size_t cols) {
  size_t row = blockIdx.x;
  size_t startInd = row * cols;
  float maxScore = -99999;
  size_t maxInd;
  for(size_t col = 0; col < cols; ++col) {
    size_t ind = startInd + col;
    float score = data[ind];
    if(score > maxScore) {
      maxScore = score;
      maxInd = col;
    }
  }
  out[row] = maxInd;
}

///////////////////////////////////////////////////////

void Prod(hipblasHandle_t handle,
          Tensor C,
          const Tensor A,
          const Tensor B,
          bool transA,
          bool transB,
          float beta,
          float scalar) {
  hipSetDevice(C->getDevice());
  float alpha = scalar;

  size_t m = A->shape().elements() / A->shape().back();
  size_t k = A->shape().back();
  if(transA)
    std::swap(m, k);

  size_t l = B->shape().elements() / B->shape().back();
  size_t n = B->shape().back();
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape().back();
  size_t ldb = B->shape().back();
  size_t ldc = B->shape().back();

  if(transB)
    ldc = B->shape().elements() / B->shape().back();

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

#if CUDA_VERSION >= 9000
  //hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
#endif
  hipblasSgemm(handle,
              opB,
              opA,
              n,
              m,
              k,
              &alpha,
              B->data(),
              ldb,
              A->data(),
              lda,
              &beta,
              C->data(),
              ldc);
#if CUDA_VERSION >= 9000
  //hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
#endif
}

void ProdBatched(hipblasHandle_t handle,
                 Tensor C,
                 const Tensor A,
                 const Tensor B,
                 bool transA,
                 bool transB,
                 float beta,
                 float scalar) {
  hipSetDevice(C->getDevice());
  float alpha = scalar;

  size_t batchA = A->shape().elements() / (A->shape()[-1] * A->shape()[-2]);
  size_t batchB = B->shape().elements() / (B->shape()[-1] * B->shape()[-2]);

  size_t m = A->shape()[-2];
  size_t k = A->shape()[-1];
  if(transA)
    std::swap(m, k);

  size_t l = B->shape()[-2];
  size_t n = B->shape()[-1];
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape()[-1];
  size_t ldb = B->shape()[-1];
  size_t ldc = B->shape()[-1];

  if(transB)
    ldc = B->shape()[-2];

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

#if CUDA_VERSION >= 9000
  //hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
#endif
  hipblasSgemmStridedBatched(handle,
                            opB,
                            opA,
                            n,
                            m,
                            k,
                            &alpha,
                            B->data(),
                            ldb,
                            batchB == 1 ? 0 : n * k,
                            A->data(),
                            lda,
                            batchA == 1 ? 0 : m * k,
                            &beta,
                            C->data(),
                            ldc,
                            n * m,
                            std::max(batchA, batchB));
#if CUDA_VERSION >= 9000
  //hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
#endif
}

__global__ void gCopyRows(float* out,
                          const float* in,
                          size_t cols,
                          const size_t* sourceRowIdx,
                          size_t rows) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      size_t dstId = j;
      size_t srcId = sourceRowIdx[j];

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

void CopyRows(Tensor out, const Tensor in, const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  size_t cols = in->shape().back();
  size_t rowsToCopy = indices.size();

  int threads = std::min(MAX_THREADS, (int)cols);
  int blocks = std::min(MAX_BLOCKS, (int)rowsToCopy);

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, rowsToCopy * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        rowsToCopy * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gCopyRows<<<blocks, threads>>>(
      out->data(), in->data(), cols, d_indices, rowsToCopy);

  CUDA_CHECK(hipFree(d_indices));
}

__global__ void gPasteRows(float* out,
                           const float* in,
                           size_t cols,
                           const size_t* targetRowIdx,
                           size_t rows) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      size_t dstId = targetRowIdx[j];
      size_t srcId = j;

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          atomicAdd(rowOut + i, rowIn[i]);
      }
    }
  }
}

void PasteRows(Tensor out,
               const Tensor in,
               const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  size_t cols = in->shape().back();
  size_t rowsToCopy = indices.size();

  int threads = std::min(MAX_THREADS, (int)cols);
  int blocks = std::min(MAX_BLOCKS, (int)rowsToCopy);

  // @TODO: turn into tensor
  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, rowsToCopy * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        rowsToCopy * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gPasteRows<<<blocks, threads>>>(
      out->data(), in->data(), cols, d_indices, rowsToCopy);
  CUDA_CHECK(hipFree(d_indices));
}

/////////////

__global__ void gCopyCols(float* out,
                          const float* in,
                          size_t rows,
                          size_t colsIn,
                          const size_t* sourceColIdx,
                          size_t colsOut) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* rowIn = in + j * colsIn;
      float* rowOut = out + j * colsOut;

      for(int tid = 0; tid < colsOut; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < colsOut)
          rowOut[i] = rowIn[sourceColIdx[i]];
      }
    }
  }
}

void CopyCols(Tensor out, const Tensor in, const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  size_t rows = in->shape().elements() / in->shape().back();
  size_t cols = in->shape().back();

  size_t colsToCopy = indices.size();

  int threads = std::min(MAX_THREADS, (int)colsToCopy);
  int blocks = std::min(MAX_BLOCKS, (int)rows);

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, colsToCopy * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        colsToCopy * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gCopyCols<<<blocks, threads>>>(
      out->data(), in->data(), rows, cols, d_indices, colsToCopy);

  CUDA_CHECK(hipFree(d_indices));
}

__global__ void gPasteCols(float* out,
                           const float* in,
                           size_t rows,
                           size_t colsOut,
                           const size_t* targetColIdx,
                           size_t colsIn) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* rowIn = in + j * colsIn;
      float* rowOut = out + j * colsOut;

      for(int tid = 0; tid < colsIn; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < colsIn)
          rowOut[targetColIdx[i]] = rowIn[i];
      }
    }
  }
}

void PasteCols(Tensor out,
               const Tensor in,
               const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  size_t rows = in->shape().elements() / in->shape().back();
  size_t cols = in->shape().back();

  size_t colsToCopy = indices.size();

  int threads = std::min(MAX_THREADS, (int)colsToCopy);
  int blocks = std::min(MAX_BLOCKS, (int)rows);

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, colsToCopy * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        colsToCopy * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gPasteCols<<<blocks, threads>>>(
      out->data(), in->data(), rows, cols, d_indices, colsToCopy);

  CUDA_CHECK(hipFree(d_indices));
}

__global__ void gSelect(float* out,
                        gpu::Shape outShape,
                        const float* in,
                        const gpu::Shape inShape,
                        int axis,
                        size_t* d_indices) {
  int length = outShape.elements();
  gpu::Array<int, gpu::Shape::size()> dims;

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      outShape.dims(index, dims);
      dims[axis] = d_indices[dims[axis]];
      int inIndex = inShape.index(dims);
      out[index] = in[inIndex];
    }
  }
}

__global__ void gInsert(float* out,
                        gpu::Shape outShape,
                        const float* in,
                        const gpu::Shape inShape,
                        int axis,
                        size_t* d_indices) {
  int length = inShape.elements();
  gpu::Array<int, gpu::Shape::size()> dims;

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      inShape.dims(index, dims);
      dims[axis] = d_indices[dims[index]];
      int outIndex = outShape.index(dims);
      out[outIndex] = in[index];
    }
  }
}

void Select(Ptr<Allocator<DeviceGPU>> allocator,
            Tensor out,
            const Tensor in,
            int axis,
            const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  auto mp_indices = allocator->alloc<size_t>(indices.size());
  mp_indices->insert(indices.data(), indices.size());

  int axisGPU = axis + gpu::Shape::size() - out->shape().size();
  gSelect<<<blocks, threads>>>(out->data(),
                               out->shape(),
                               in->data(),
                               in->shape(),
                               axisGPU,
                               mp_indices->data<size_t>());

  allocator->free(mp_indices);
}

void Insert(Ptr<Allocator<DeviceGPU>> allocator,
            Tensor out,
            const Tensor in,
            int axis,
            const std::vector<size_t>& indices) {
  hipSetDevice(in->getDevice());

  int length = in->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  auto mp_indices = allocator->alloc<size_t>(indices.size());
  mp_indices->insert(indices.data(), indices.size());

  int axisGPU = axis + gpu::Shape::size() - out->shape().size();
  gInsert<<<blocks, threads>>>(out->data(),
                               out->shape(),
                               in->data(),
                               in->shape(),
                               axisGPU,
                               mp_indices->data<size_t>());

  allocator->free(mp_indices);
}

__global__ void gGRUFastForward(float* out,
                                const float* state,
                                const float* xW,
                                const float* sU,
                                const float* b,
                                const float* mask,
                                size_t rows,
                                size_t cols,
                                bool final) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];
      float* rowOut = out + j * cols;
      const float* rowState = state + j * cols;

      const float* xWrow = xW + j * cols * 3;
      const float* sUrow = sU + j * cols * 3;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          float r = stableLogit(xWrow[i] + sUrow[i] + b[i]);

          int k = i + cols;

          float z = stableLogit(xWrow[k] + sUrow[k] + b[k]);

          int l = i + 2 * cols;
          float h;
          if(final)
            h = tanhf(xWrow[l] + (sUrow[l] + b[l]) * r);
          else
            h = tanhf(xWrow[l] + sUrow[l] * r + b[l]);

          float out = (1.0f - z) * h + z * rowState[i];
          rowOut[i] = m * out + (1 - m) * rowState[i];
        }
      }
    }
  }
}

void GRUFastForward(Tensor out, std::vector<Tensor> inputs, bool final) {
  hipSetDevice(out->getDevice());

  int rows = out->shape().elements() / out->shape().back();
  int cols = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gGRUFastForward<<<blocks, threads>>>(
      out->data(),                                // output
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      rows,
      cols,
      final);
}

__global__ void gGRUFastBackward(float* outState,
                                 float* outXW,
                                 float* outSU,
                                 float* outB,
                                 const float* state,
                                 const float* xW,
                                 const float* sU,
                                 const float* b,
                                 const float* mask,
                                 const float* adj,
                                 size_t rows,
                                 size_t cols,
                                 bool final) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOutState = outState + j * cols;
      float* rowOutXW = outXW + j * cols * 3;
      float* rowOutSU = outSU + j * cols * 3;

      const float* rowState = state + j * cols;
      const float* rowXW = xW + j * cols * 3;
      const float* rowSU = sU + j * cols * 3;
      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          int k = i + cols;
          int l = i + 2 * cols;

          float r = stableLogit(rowXW[i] + rowSU[i] + b[i]);
          float z = stableLogit(rowXW[k] + rowSU[k] + b[k]);

          float h;
          if(final)
            h = tanhf(rowXW[l] + (rowSU[l] + b[l]) * r);
          else
            h = tanhf(rowXW[l] + rowSU[l] * r + b[l]);

          float adj = rowAdj[i];

          float t = (1 - z) * (1 - h * h);

          // df/ds
          if(outState)
            rowOutState[i] += (m * z - m + 1) * adj;

          // df/d(xW_r) ...
          float dfdxW_r = m * r * (1 - r) * t * adj;
          if(final)
            dfdxW_r *= rowSU[l] + b[l];
          else
            dfdxW_r *= rowSU[l];
          if(outXW)
            rowOutXW[i] += dfdxW_r;
          if(outSU)
            rowOutSU[i] += dfdxW_r;
          if(outB)
            atomicAdd(outB + i, dfdxW_r);

          // df/d(xW_z) ...
          float dfdxW_z = m * (1 - z) * z * (rowState[i] - h) * adj;
          if(outXW)
            rowOutXW[k] += dfdxW_z;
          if(outSU)
            rowOutSU[k] += dfdxW_z;
          if(outB)
            atomicAdd(outB + k, dfdxW_z);

          // df/d(xW_x) ...
          float dfdxW_x = m * t * adj;
          if(outXW)
            rowOutXW[l] += dfdxW_x;
          if(outSU)
            rowOutSU[l] += dfdxW_x * r;
          if(outB)
            if(final)
              atomicAdd(outB + l, dfdxW_x * r);
            else
              atomicAdd(outB + l, dfdxW_x);
        }
      }
    }
  }
}

void GRUFastBackward(std::vector<Tensor> outputs,
                     std::vector<Tensor> inputs,
                     Tensor adj,
                     bool final) {
  hipSetDevice(adj->getDevice());

  int rows = adj->shape().elements() / adj->shape().back();
  int cols = adj->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gGRUFastBackward<<<blocks, threads>>>(
      outputs[0] ? outputs[0]->data() : 0,        // state - adj
      outputs[1] ? outputs[1]->data() : 0,        // xW - adj
      outputs[2] ? outputs[2]->data() : 0,        // sU - adj
      outputs[3] ? outputs[3]->data() : 0,        // b - adj
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      adj->data(),
      rows,
      cols,
      final);
}

__global__ void gCrossEntropyPick(float* out,
                                  const gpu::Shape outShape,
                                  const float* in,
                                  const gpu::Shape inShape,
                                  const float* pick) {
  int rows = inShape.elements() / inShape.back();
  int cols = inShape.back();

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* sp = in + j * cols;

      extern __shared__ float _share[];
      float* _max = _share + blockDim.x;

      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if(sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += __expf(sp[id] - max);
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // cross-entropy
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id == (int)pick[j]) {
          out[j] = __logf(_sum[0]) - sp[id] + max;
        }
      }
    }
  }
}

void CrossEntropyPick(Tensor out, Tensor in, Tensor pick) {
  hipSetDevice(out->getDevice());

  int rows = in->shape().elements() / in->shape().back();
  int cols = in->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)rows);
  int threads = std::min(MAX_THREADS, (int)cols);
  int shared = sizeof(float) * threads * 2;

  gCrossEntropyPick<<<blocks, threads, shared>>>(
      out->data(), out->shape(), in->data(), in->shape(), pick->data());
}

__global__ void gCrossEntropyPickBackward(float* out,
                                          const gpu::Shape outShape,
                                          const float* adj,
                                          const float* in,
                                          const float* pick) {
  int rows = outShape.elements() / outShape.back();
  int cols = outShape.back();
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* sp = in + j * cols;
      float* so = out + j * cols;

      extern __shared__ float _share[];
      float* _max = _share + blockDim.x;

      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if(sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = __expf(sp[id] - max);
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // cross-entropy
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float sub = (float)(id == (int)pick[j]);
          so[id] += adj[j] * (__expf(sp[id] - max) / _sum[0] - sub);
        }
      }
    }
  }
}

void CrossEntropyPickBackward(Tensor out, Tensor adj, Tensor a, Tensor pick) {
  hipSetDevice(out->getDevice());

  int rows = out->shape().elements() / out->shape().back();
  int cols = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)rows);
  int threads = std::min(MAX_THREADS, (int)cols);
  int shared = sizeof(float) * threads * 2;

  gCrossEntropyPickBackward<<<blocks, threads, shared>>>(
      out->data(), out->shape(), adj->data(), a->data(), pick->data());
}


float L2Norm(Tensor in) {
  using namespace functional;

  hipSetDevice(in->getDevice());

  int size = in->shape().elements();
  int threads = std::min(MAX_THREADS, size);
  int blocks  = std::min(MAX_BLOCKS, size / threads  + (size % threads != 0));

  uint8_t* data;
  hipMalloc(&data, blocks * sizeof(float));
  Tensor out(new TensorBase(
      New<MemoryPiece>(data, blocks * sizeof(float)), {1, blocks}, in->getDevice()));

  ReduceAll(_1 * _1, out, in);
  float dataCpu = sqrtf(out->get(0));
  out.reset();
  hipFree(data);
  return dataCpu;
}

__global__ void gAtt(float* out,
                     const float* va,
                     const float* ctx,
                     const float* state,
                     int m,  // total rows (batch x time x beam)
                     int k,  // depth
                     int b,  // batch size
                     int t  // time of ctx
                     ) {
  int rows = m;
  int cols = k;

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* vaRow = va;
      const float* ctxRow = ctx + (j % (b * t)) * cols;
      const float* stateRow = state + ((j / (b * t)) * b + j % b) * cols;

      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float z = ctxRow[id] + stateRow[id];
          float ex = tanhf(z) * vaRow[id];
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      out[j] = _sum[0];
    }
  }
}

void Att(Tensor out, Tensor va, Tensor context, Tensor state) {
  hipSetDevice(out->getDevice());

  size_t m = out->shape().elements() / out->shape().back();

  size_t dims = context->shape().size();
  size_t k = context->shape()[dims - 1];
  size_t b = context->shape()[dims - 2];
  size_t t = context->shape()[dims - 3];

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads * 2;

  gAtt<<<blocks, threads, shared>>>(out->data(),
                                    va->data(),
                                    context->data(),
                                    state->data(),
                                    m,
                                    k,
                                    b,
                                    t);
}

__global__ void gAttBack(float* gVa,
                         float* gContext,
                         float* gState,
                         const float* va,
                         const float* context,
                         const float* state,
                         const float* adj,
                         int m,  // rows
                         int k,  // cols
                         int n   // batch size
                         ) {
  int rows = m;
  int cols = k;
  for(int bid = 0; bid < m; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* gcRow = gContext + j * cols;
      float* gsRow = gState + (j % n) * cols;

      const float* cRow = context + j * cols;
      const float* sRow = state + (j % n) * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float z = cRow[id] + sRow[id];

          float t = tanhf(z);
          float r = va[id] * (1.f - t * t);

          gcRow[id] += r * adj[j];
          gsRow[id] += r * adj[j];
          atomicAdd(gVa + id, t * adj[j]);
        }
      }
    }
  }
}

void AttBack(Tensor gVa,
             Tensor gContext,
             Tensor gState,
             Tensor va,
             Tensor context,
             Tensor state,
             Tensor adj) {
  hipSetDevice(adj->getDevice());

  size_t m = adj->shape().elements() / adj->shape().back();

  size_t dims = context->shape().size();
  size_t k = context->shape()[dims - 1];
  size_t n = context->shape()[dims - 2];

  int blocks = std::min(MAX_BLOCKS, (int)n);
  int threads = std::min(MAX_THREADS, (int)k);

  gAttBack<<<blocks, threads>>>(gVa->data(),
                                gContext->data(),
                                gState->data(),

                                va->data(),
                                context->data(),
                                state->data(),

                                adj->data(),
                                m,
                                k,
                                n);
}

__global__ void gLNormalization(float* out,
                                const float* in,
                                const float* alpha,
                                const float* beta,
                                int rows,
                                int cols,
                                float eps = 1e-9) {
  extern __shared__ float _share[];

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0f;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float mean = _sum[0] / cols;
      __syncthreads();

      float* _sqSum = _share + blockDim.x;

      _sqSum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = sp[id] - mean;
          _sqSum[threadIdx.x] += ex * ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sqSum[threadIdx.x] += _sqSum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float sigma = sqrtf(eps + (_sqSum[0] / cols));
      __syncthreads();

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float t = alpha[id] * ((sp[id] - mean) / sigma);
          if(beta != nullptr)
            t += beta[id];
          so[id] = t;
        }
      }
    }
  }
}

void LayerNormalization(Tensor out,
                        Tensor in,
                        Tensor gamma,
                        Tensor beta,
                        float eps) {
  hipSetDevice(out->getDevice());

  int rows = in->shape().elements() / in->shape().back();
  int cols = in->shape().back();

  int blocks = std::min(MAX_BLOCKS, (int)rows);
  int threads = std::min(MAX_THREADS, (int)cols);
  int shared = 2 * threads * sizeof(float);

  gLNormalization<<<blocks, threads, shared>>>(out->data(),
                                               in->data(),
                                               gamma->data(),
                                               beta ? beta->data() : nullptr,
                                               rows,
                                               cols,
                                               eps);
}

__global__ void gLayerNormalizationGrad(float* gradX,
                                        float* gradGamma,
                                        float* gradBeta,
                                        float* adj,
                                        float* y,
                                        float* x,
                                        float* gamma,
                                        float* beta,
                                        int rows,
                                        int cols,
                                        float eps = 1e-9) {
  extern __shared__ float shared[];

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* sum_adj = shared;
      float* sum_adj_x = shared + blockDim.x;
      float* sum_x = shared + 2 * blockDim.x;
      float* sum_sqr = shared + 3 * blockDim.x;

      const float* xRow = x + j * cols;
      const float* yRow = y + j * cols;
      const float* adjRow = adj + j * cols;
      float* gradXRow = gradX + j * cols;

      sum_x[threadIdx.x] = 0.0f;
      sum_adj[threadIdx.x] = 0.0f;
      sum_adj_x[threadIdx.x] = 0.0f;
      sum_sqr[threadIdx.x] = 0.0f;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          sum_x[threadIdx.x] += xRow[id];
          sum_adj_x[threadIdx.x]
              += adjRow[id] * (yRow[id] - ((beta) ? beta[id] : 0)) / gamma[id];
          sum_adj[threadIdx.x] += adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          sum_x[threadIdx.x] += sum_x[threadIdx.x + skip];
          sum_adj[threadIdx.x] += sum_adj[threadIdx.x + skip];
          sum_adj_x[threadIdx.x] += sum_adj_x[threadIdx.x + skip];
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float mean = sum_x[0] / cols;
      __syncthreads();

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = xRow[id] - mean;
          sum_sqr[threadIdx.x] += ex * ex;
        }
      }

      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          sum_sqr[threadIdx.x] += sum_sqr[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float sigma = sqrtf(eps + (sum_sqr[0] / cols));
      __syncthreads();

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float grad_x = 0.0f;
          float x_hat = (yRow[id] - ((beta) ? beta[id] : 0)) / gamma[id];
          grad_x += cols * adjRow[id];
          grad_x -= sum_adj[0];
          grad_x -= sum_adj_x[0] * x_hat;
          grad_x /= (cols * sigma);

          float valX = gamma[id] * grad_x;
          float sign = (0.f < valX) - (valX < 0.f);
          valX = fabs(valX) > 1000 ? sign * 1000 : valX;

          gradXRow[id] += valX;
          atomicAdd(gradGamma + id, adjRow[id] * x_hat);
          if(beta) {
            atomicAdd(gradBeta + id, adjRow[id]);
          }
        }
      }
    }
  }
}

void LayerNormalizationGrad(Tensor gradX,
                            Tensor gradGamma,
                            Tensor gradBeta,
                            Tensor adj,
                            Tensor y,
                            Tensor x,
                            Tensor gamma,
                            Tensor beta,
                            float eps) {
  hipSetDevice(adj->getDevice());
  int rows = y->shape().elements() / y->shape().back();
  int cols = y->shape().back();

  int threads = std::min(MAX_THREADS, cols);
  int blocks = std::min(MAX_BLOCKS, rows);
  int shared = sizeof(float) * threads * 4;

  gLayerNormalizationGrad<<<blocks, threads, shared>>>(
      gradX->data(),
      gradGamma->data(),
      (gradBeta) ? gradBeta->data() : nullptr,
      adj->data(),
      y->data(),
      x->data(),
      gamma->data(),
      (beta) ? beta->data() : nullptr,
      rows,
      cols,
      eps);
}

__global__ void gShift(float* out, const float* in, int length, int offset) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      if(index - offset < 0 || index - offset >= length)
        out[index] = 0;
      else
        out[index] = in[index - offset];
    }
  }
}

void Shift(Tensor out, Tensor in, Shape shift, bool invert) {

  UTIL_THROW_IF2(in->shape().size() != shift.size(), "bad dimensions");

  int offset = 0;
  for(int i = 0; i < shift.size(); ++i)
    offset += in->shape().stride(i) * shift[i];

  if(invert)
    offset = -offset;

  hipSetDevice(out->getDevice());

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gShift<<<blocks, threads>>>(out->data(), in->data(), length, offset);
}

__global__ void gSetSparse(float* out,
                           const size_t* indices,
                           const float* values,
                           int length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      out[indices[index]] = values[index];
    }
  }
}

void SetSparse(float* out,
               const std::vector<size_t>& indices,
               const std::vector<float>& values) {
  int length = indices.size();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, length * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        length * sizeof(size_t),
                        hipMemcpyHostToDevice));

  float* d_values;
  CUDA_CHECK(hipMalloc(&d_values, length * sizeof(float)));
  CUDA_CHECK(hipMemcpy(
      d_values, values.data(), length * sizeof(float), hipMemcpyHostToDevice));

  gSetSparse<<<blocks, threads>>>(out, d_indices, d_values, length);

  hipFree(d_indices);
  hipFree(d_values);
}

/******************************************************************************/

__global__ void gLSTMCellForward(float* out,
                                 const float* cell,
                                 const float* xW,
                                 const float* sU,
                                 const float* b,
                                 const float* mask,
                                 size_t rows,
                                 size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOut = out + j * cols;
      const float* rowCell = cell + j * cols;

      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          float gf = stableLogit(xWrow[i] + sUrow[i] + b[i]);

          int k = i + cols;
          float gi = stableLogit(xWrow[k] + sUrow[k] + b[k]);

          int l = i + 2 * cols;
          float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);

          float cout = gf * rowCell[i] + gi * gc;
          rowOut[i] = m * cout + (1 - m) * rowCell[i];
        }
      }
    }
  }
}

void LSTMCellForward(Tensor out, std::vector<Tensor> inputs) {
  hipSetDevice(out->getDevice());

  int rows = out->shape().elements() / out->shape().back();
  int cols = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMCellForward<<<blocks, threads>>>(
      out->data(),                                // output
      inputs[0]->data(),                          // cell state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      rows,
      cols);
}

__global__ void gLSTMOutputForward(float* out,
                                   const float* cell,
                                   const float* xW,
                                   const float* sU,
                                   const float* b,
                                   size_t rows,
                                   size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols;
      const float* rowCell = cell + j * cols;

      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          int k = i + 3 * cols;
          float go = stableLogit(xWrow[k] + sUrow[k] + b[k]);

          rowOut[i] = go * tanhf(rowCell[i]);
        }
      }
    }
  }
}

void LSTMOutputForward(Tensor out, std::vector<Tensor> inputs) {
  hipSetDevice(out->getDevice());

  int rows = out->shape().elements() / out->shape().back();
  int cols = out->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMOutputForward<<<blocks, threads>>>(out->data(),        // output
                                          inputs[0]->data(),  // cell state
                                          inputs[1]->data(),  // xW
                                          inputs[2]->data(),  // sU
                                          inputs[3]->data(),  // b
                                          rows,
                                          cols);
}

__global__ void gLSTMCellBackward(float* outCell,
                                  float* outXW,
                                  float* outSU,
                                  float* outB,
                                  const float* cell,
                                  const float* xW,
                                  const float* sU,
                                  const float* b,
                                  const float* mask,
                                  const float* adj,
                                  size_t rows,
                                  size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOutCell = outCell + j * cols;
      float* rowOutXW = outXW + j * cols * 4;
      float* rowOutSU = outSU + j * cols * 4;

      const float* rowCell = cell + j * cols;
      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          float gf = stableLogit(xWrow[i] + sUrow[i] + b[i]);

          int k = i + cols;
          float gi = stableLogit(xWrow[k] + sUrow[k] + b[k]);

          int l = i + 2 * cols;
          float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);

          float adj = rowAdj[i];

          // dc/dc_{t-1}
          if(outCell)
            rowOutCell[i] += (m * gf - m + 1) * adj;

          // dc/d(b_f) = dc/d(xW_f) ...
          float dcdxf = m * rowCell[i] * gf * (1 - gf) * adj;
          if(outXW)
            rowOutXW[i] += dcdxf;
          if(outSU)
            rowOutSU[i] += dcdxf;
          if(outB)
            atomicAdd(outB + i, dcdxf);

          // dc/d(b_i) ...
          float dcdb_i = m * gc * gi * (1 - gi) * adj;
          if(outXW)
            rowOutXW[k] += dcdb_i;
          if(outSU)
            rowOutSU[k] += dcdb_i;
          if(outB)
            atomicAdd(outB + k, dcdb_i);

          // dc/d(b_c) ...
          float dcdxc = m * gi * (1 - gc * gc) * adj;
          if(outXW)
            rowOutXW[l] += dcdxc;
          if(outSU)
            rowOutSU[l] += dcdxc;
          if(outB)
            atomicAdd(outB + l, dcdxc);
        }
      }
    }
  }
}

void LSTMCellBackward(std::vector<Tensor> outputs,
                      std::vector<Tensor> inputs,
                      Tensor adj) {
  hipSetDevice(adj->getDevice());

  int rows = adj->shape().elements() / adj->shape().back();
  int cols = adj->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMCellBackward<<<blocks, threads>>>(
      outputs[0] ? outputs[0]->data() : 0,        // state - adj
      outputs[1] ? outputs[1]->data() : 0,        // xW - adj
      outputs[2] ? outputs[2]->data() : 0,        // sU - adj
      outputs[3] ? outputs[3]->data() : 0,        // b - adj
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      adj->data(),
      rows,
      cols);
}

__global__ void gLSTMOutputBackward(float* outCell,
                                    float* outXW,
                                    float* outSU,
                                    float* outB,
                                    const float* cell,
                                    const float* xW,
                                    const float* sU,
                                    const float* b,
                                    const float* adj,
                                    size_t rows,
                                    size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOutCell = outCell + j * cols;
      float* rowOutXW = outXW + j * cols * 4;
      float* rowOutSU = outSU + j * cols * 4;

      const float* rowCell = cell + j * cols;
      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          int k = i + 3 * cols;
          float go = stableLogit(xWrow[k] + sUrow[k] + b[k]);

          float t = tanhf(rowCell[i]);

          float adj = rowAdj[i];

          // dc/dc_{t-1}
          if(outCell)
            rowOutCell[i] += go * (1 - t * t) * adj;

          // dc/d(b_o) = dc/d(xW_f) ...
          float dcdxo = t * go * (1 - go) * adj;
          if(outXW)
            rowOutXW[k] += dcdxo;
          if(outSU)
            rowOutSU[k] += dcdxo;
          if(outB)
            atomicAdd(outB + k, dcdxo);
        }
      }
    }
  }
}

void LSTMOutputBackward(std::vector<Tensor> outputs,
                        std::vector<Tensor> inputs,
                        Tensor adj) {
  hipSetDevice(adj->getDevice());

  int rows = adj->shape().elements() / adj->shape().back();
  int cols = adj->shape().back();

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMOutputBackward<<<blocks, threads>>>(
      outputs[0] ? outputs[0]->data() : 0,  // state - adj
      outputs[1] ? outputs[1]->data() : 0,  // xW - adj
      outputs[2] ? outputs[2]->data() : 0,  // sU - adj
      outputs[3] ? outputs[3]->data() : 0,  // b - adj
      inputs[0]->data(),                    // state
      inputs[1]->data(),                    // xW
      inputs[2]->data(),                    // sU
      inputs[3]->data(),                    // b
      adj->data(),
      rows,
      cols);
}

__global__ void gHighwayForward(float* out,
                                const float* in1,
                                const float* in2,
                                const float* t,
                                size_t length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      float sigma = stableLogit(t[index]);
      out[index] = in1[index] * sigma + in2[index] * (1.f - sigma);
    }
  }
}

void HighwayForward(Tensor out,
                    const Tensor in1,
                    const Tensor in2,
                    const Tensor t) {
  hipSetDevice(out->getDevice());

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gHighwayForward<<<blocks, threads>>>(
      out->data(), in1->data(), in2->data(), t->data(), length);
}

__global__ void gHighwayBackward(float* out1,
                                 float* out2,
                                 float* outt,
                                 const float* in1,
                                 const float* in2,
                                 const float* t,
                                 const float* adj,
                                 size_t length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      float sigma = stableLogit(t[index]);
      out1[index] = sigma * adj[index];
      out2[index] = (1.f - sigma) * adj[index];
      outt[index]
          = sigma * (1.f - sigma) * (in1[index] - in2[index]) * adj[index];
    }
  }
}

void HighwayBackward(Tensor out1,
                     Tensor out2,
                     Tensor outt,
                     const Tensor in1,
                     const Tensor in2,
                     const Tensor t,
                     const Tensor adj) {
  hipSetDevice(out1->getDevice());

  int length = out1->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gHighwayBackward<<<blocks, threads>>>(out1->data(),
                                        out2->data(),
                                        outt->data(),
                                        in1->data(),
                                        in2->data(),
                                        t->data(),
                                        adj->data(),
                                        length);
}

__global__ void gMaxPoolingForward(float* out,
                                   int outRows,
                                   int outCols,
                                   float* in,
                                   int inRows,
                                   int inCols,
                                   float* mask,
                                   int numKernels,
                                   int maskCols,
                                   int width,
                                   int lastWidth) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= outRows * outCols) return;

  int rowId = tid / outRows;
  int colId = tid % outRows;

  float* b = in + (rowId * inCols) + (colId * width);

  if (colId == outRows - 1) {
    width = lastWidth;
  }

  float* localMask = mask  + (rowId / numKernels) * maskCols + colId * width;
  float currentMax = b[0] * localMask[0];
  for (int i = 1; i < width; ++i) {
    if (b[i] * localMask[i] > currentMax) {
      currentMax = b[i] * localMask[i];
    }
  }

  out[rowId + (colId * outCols)] = currentMax;
}

void PoolingWithMaskingForward(Tensor out,
                               Tensor in,
                               Tensor mask,
                               int width,
                               bool isEven) {
  int n = out->shape().elements();
  int threads = std::min(n, MAX_THREADS);
  int blocks = n / threads + (n % threads != 0);

  Shape& inShape = in->shape();
  int inRows = inShape[0] * inShape[1];
  int inCols = inShape[2];

  Shape& outShape = out->shape();
  int outRows = outShape[2];
  int outCols = outShape[0] * outShape[1];

  int lastWidth = ((inCols - isEven) % width == 0)
                  ? width
                  : (inCols - isEven) % width;

  gMaxPoolingForward<<<blocks, threads>>>(
      out->data(), outRows, outCols,
      in->data(), inRows, inCols,
      mask->data(), outShape[1], mask->shape()[2],
      width, lastWidth);
}

__global__ void gMaxPoolingBackward(float* adj,
                                    int adjRows,
                                    int adjCols,
                                    float* in,
                                    float* adjIn,
                                    int inRows,
                                    int inCols,
                                    float* mask,
                                    int numKernels,
                                    int maskCols,
                                    int width,
                                    int lastWidth)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= adjRows * adjCols) return;

  int rowId = tid / adjRows;
  int colId = tid % adjRows;

  float* b = in + (rowId * inCols) + (colId * width);

  if (colId == adjRows - 1) {
    width = lastWidth;
  }

  float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;
  size_t currentMaxIdx = 0;
  for (int i = 1; i < width; ++i) {
    if (b[i] * localMask[i] > b[currentMaxIdx] * localMask[currentMaxIdx]) {
      currentMaxIdx = i;
    }
  }

  adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx] += adj[rowId + (colId * adjCols)];
}

void PoolingWithMaskingBackward(Tensor adj,
                                Tensor adjIn,
                                Tensor in,
                                Tensor mask,
                                int width,
                                bool isEven) {
  int n = adj->shape().elements();
  int threads = std::min(n, 512);
  int blocks = n / threads + (n % threads != 0);

  Shape& inShape = in->shape();
  int inRows = inShape[0] * inShape[1];
  int inCols = inShape[2];

  Shape& adjShape = adj->shape();
  int adjRows = adjShape[2];
  int adjCols = adjShape[0] * adjShape[1];

  int lastWidth = ((inCols - isEven) % width == 0)
                  ? width
                  : (inCols - isEven) % width;

  gMaxPoolingBackward<<<blocks, threads>>>(
      adj->data(), adjRows, adjCols,
      in->data(), adjIn->data(), inRows, inCols,
      mask->data(), adjShape[1], mask->shape()[2],
      width, lastWidth);
}

}  // namespace marian
